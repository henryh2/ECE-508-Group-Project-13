#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define CPU_MODE 0
#define GPU_MODE 1

#define BLOCK_SIZE 512

void local_clustering_coefficent(const pangolin::COOView<uint32_t> view,    // graph
                                 uint64_t* coefficents,                     // coefficients array
                                 int numNodes                               // number of nodes in the graph
) {
  dim3 dimBlock(BLOCK_SIZE);
  // calculate the number of blocks needed
  dim3 dimGridCount(ceil(view.nnz() * 1.0 / BLOCK_SIZE));

  // Store triangle counts for each node 
  uint64_t *triangleCounts;
  hipMalloc((void**) (&triangleCounts), numNodes * sizeof(uint64_t));
  // Kernel coefficients array
  float *kernel_coefficents;
  hipMalloc((void**) (&kernel_coefficents), numNodes * sizeof(float));

  triangle_count_kernel<<<dimGridCount, dimBlock>>>(triangleCounts, view.row_ind(), view.col_ind(), view.row_ptr(), view.nnz());

  // One thread calculate the coefficient for one node
  dim3 dimGridCoefficient(numNodes * 1.0 / BLOCK_SIZE)
  // launch another kernal to compute llc
  coefficients_calculate_kernel<<<dimGridCoefficient, dimBlock>>>(triangleCounts, view.row_ptr(), kernel_coefficents, numNodes);

  cudaMemCpy(coefficients, kernel_coefficents, sizeof(float) * numNodes, cudaMemCpyDeviceToHost);

  hipFree(kernel_coefficents);
  hipFree(triangleCounts);
}

/**
 * find number of intersections using binary search
 * uPtr is linear pointer
 * vPtr is binary pointer
*/
__device__ static uint64_t binary_search(const uint32_t *const edgeDst, uint32_t uPtr, uint32_t uEnd, uint32_t vPtr, uint32_t vEnd) {
  uint64_t tc = 0;
  while(uPtr < uEnd) {
    uint32_t w1 = edgeDst[uPtr];
    int left = vPtr;
    int right = vEnd;

    while(left < right) {
      int mid = left + (right - left) / 2;
      // int mid = (right - left) / 2;
      uint32_t w2 = edgeDst[mid];
      if(w1 > w2) {
        left = mid + 1;
      }
      else if(w1 < w2) {
        right = mid;
      }
      else {
        tc++;
        break;
      }
    }
    uPtr ++;
  }

  return tc;
}

/**
 * find number of intersections using lienar search
*/
__device__ static uint64_t linear_search(const uint32_t *const edgeDst, uint32_t uPtr, uint32_t uEnd, uint32_t vPtr, uint32_t vEnd) {
  uint64_t tc = 0;
  while(uPtr < uEnd && vPtr < vEnd) {
    uint32_t w1 = edgeDst[uPtr];
    uint32_t w2 = edgeDst[vPtr];
    if(w1 < w2) {
      uPtr++;
    }
    else if(w1 > w2) {
      vPtr++;
    }
    else {
      uPtr++;
      vPtr++;
      tc ++;
    }
  }
  return tc;
}

__global__ static void triangle_count_kernel(uint64_t *__restrict__ triangleCounts, //!< per-node triangle counts
                                 const uint32_t *const edgeSrc,         //!< node ids for edge srcs
                                 const uint32_t *const edgeDst,         //!< node ids for edge dsts
                                 const uint32_t *const rowPtr,          //!< source node offsets in edgeDst
                                 const size_t numEdges                  //!< how many edges to count triangles for
) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  // The source node number
  int nodeNum = edgeSrc[tx];
  if(tx < numEdges) {
    uint32_t uPtr = rowPtr[nodeNum];
    uint32_t uEnd = rowPtr[nodeNum + 1];
    uint32_t vPtr = rowPtr[edgeDst[tx]];
    uint32_t vEnd = rowPtr[edgeDst[tx] + 1];

    uint32_t uDiff = uEnd - uPtr;
    uint32_t vDiff = vEnd - vPtr;

    // From triangle counting lab
    // using binary search when V was as least 64 and V/U was at least 6 (V is the longer list length, and U the shorter one).
    if (uDiff > vDiff && uDiff >= 64 && uDiff / vDiff >= 6) {
      // One node may have many edges, use atomic add
      atomicAdd(&triangleCounts[nodeNum], binary_search(edgeDst, vPtr, vEnd, uPtr, uEnd));
    }
    else if(vDiff > uDiff && vDiff >= 64 && vDiff / uDiff >= 6) {
      atomicAdd(&triangleCounts[nodeNum], binary_search(edgeDst, vPtr, vEnd, uPtr, uEnd));
    }
    else{
      atomicAdd(&triangleCounts[nodeNum], linear_search(edgeDst, vPtr, vEnd, uPtr, uEnd));
    }
  }
  __syncthreads();
    
}


__global__ static void coefficients_calculate_kernel(uint64_t *__restrict__ triangleCounts,     // per-node triangle counts
                                                    const uint32_t *const rowPtr,               // source node offsets in edgeDst
                                                    float *coefficients,                        // coefficients
                                                    int numNodes                                // number of nodes
) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  if(tx < numNodes) {
    int outEdge = rowPtr[tx + 1] - rowPtr[tx];
    // If the node has at least two neighbors, calculate the coefficient. Otherwise the coefficient is 0
    if(outEdge > 1) {
      coefficients[tx] = 2 * triangleCounts[tx] / (outEdge * (outEdge - 1));
    }
  }
}
